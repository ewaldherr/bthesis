
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void initializePriorities(float* priorities,hiprandState * d_state) {
    hiprand_init(1234, threadIdx.x, 0, &d_state[threadIdx.x]);
    priorities[threadIdx.x] = hiprand_uniform(d_state + threadIdx.x);
}

__global__ void checkMax(int* graph,float* priorities,int* state, int n){
    if (state[threadIdx.x] != 0) return;
    bool isMaxPriority = true;
        for (int j = 0; j < n; ++j) {
            if (graph[threadIdx.x+j*n] == 1 && state[j] == 0 && priorities[j] >= priorities[threadIdx.x]) {
                isMaxPriority = false;
                break;
            }
        }

        if (isMaxPriority) {
                state[threadIdx.x] = 1;
        }
}

__global__ void removeVertices(int* graph,int* state,bool* changes, int n){
    if (state[threadIdx.x] == 1) {
        state[threadIdx.x] = 2;
        for (int j = 0; j < n; ++j) {
            if (graph[threadIdx.x+j*n] == 1) {
                state[j] = -1;
            }
        }
        changes[0] = true; // If any vertex is added, flag a change
    }
}


// Luby's Algorithm with Kokkos
int* lubysAlgorithm(int* graph,float* priorities,int* state, int n) {
    int* host_adj = new int [n*n];
    hipMemcpy(host_adj,graph,n*n*sizeof(int),hipMemcpyDeviceToHost);
    for (int i=0;i<n;++i){
        std::cout << std::endl;
        for(int j=0;j<n;++j){
            std::cout << host_adj [i+j*n] << " ";
        }
    }
    std::cout << std::endl;
    float* host_prios = new float[n];
    int* host_state = new int[n];
    int* independentSet = new int[n];
    bool* changes = new bool[1];
    bool* d_changes = new bool[1];
    hipMalloc(&d_changes, sizeof(bool));
    hiprandState *d_state;
    hipMalloc(&d_state, sizeof(hiprandState));
    int iters = 0;
    do {
        // Step 1: Assign random priorities to remaining vertices
        initializePriorities<<<1,n>>>(priorities,d_state);
        hipMemcpy(host_prios,priorities,n*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(host_state,state,n*sizeof(int),hipMemcpyDeviceToHost);
        for (int i = 0; i< n; ++i){
            	std::cout << host_prios[i] << " " << host_state[i] << "  ";
        }
        std::cout << std::endl;
        checkMax<<<1,n>>>(graph,priorities,state,n);
        hipMemcpy(host_state,state,n*sizeof(int),hipMemcpyDeviceToHost);
        for (int i = 0; i< n; ++i){
            	std::cout << host_prios[i] << " " << host_state[i] << "  ";
        }
        std::cout << std::endl;
        // Step 3: Add selected vertices to MIS and remove them and their neighbors
        changes[0] = false;
        hipMemcpy(d_changes,changes,sizeof(bool),hipMemcpyHostToDevice);
        removeVertices<<<1,n>>>(graph,state,changes,n);
        hipMemcpy(host_state,state,n*sizeof(int),hipMemcpyDeviceToHost);
        for (int i = 0; i< n; ++i){
            	std::cout << host_prios[i] << " " << host_state[i] << "  ";
        }
        std::cout << std::endl;
        hipMemcpy(changes,d_changes,sizeof(bool),hipMemcpyDeviceToHost);
        ++ iters;
    } while (changes[0]);
    std::cout << iters << std::endl;
    hipMemcpy(independentSet,state, n*sizeof(int), hipMemcpyDeviceToHost);
    return independentSet;
}

int main(int argc, char* argv[]) {
    {
        //Initialize graph
        int n = 6;
        #define N n;
        int* adj = new int[n*n];
        for(int i = 0;i < n; ++i){
            for(int j = 0;j < n; ++j){
                adj[i+j*n] = 0;
            }
        }
        adj[0 + 1 * 6] = 1;
        adj[0 + 2 * 6] = 1;
        adj[1 + 3 * 6] = 1;
        adj[2 + 3 * 6] = 1;
        adj[3 + 4 * 6] = 1;
        adj[3 + 5 * 6] = 1;
        //backward edges
        adj[1 + 0 * 6] = 1;
        adj[2 + 0 * 6] = 1;
        adj[3 + 1 * 6] = 1;
        adj[3 + 2 * 6] = 1;
        adj[4 + 3 * 6] = 1;
        adj[5 + 3 * 6] = 1;
        for (int i=0;i<n;++i){
        std::cout << std::endl;
        for(int j=0;j<n;++j){
            std::cout << adj [i+j*n] << " ";
        }
    }
        // Run Luby's algorithm with Kokkos
        int (*d_adj)[n];
        int* host_state = new int[n];
        for(int i = 0; i < n; ++i){
            host_state[i] = 0;
        }
        int* state = new int[n];
        float* priorities = new float[n];
        int* independentSet = new int[n];
        hipMalloc(&state,n*sizeof(int));
        hipMalloc(&priorities,n*sizeof(float));
        hipMalloc((void**)&d_adj,n*n*sizeof(int));
        hipMemcpy(state,host_state,n*sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(d_adj,adj,n*n*sizeof(int),hipMemcpyHostToDevice);
        independentSet = lubysAlgorithm(adj,priorities,state,n);
        hipFree(state);
        hipFree(priorities);
        hipFree(d_adj);
        // Print the result
        std::cout << "Maximum Independent Set (MIS) nodes: " << std::endl;
        for(int i = 0; i < n; ++i){
            if (independentSet[i] == 2) {
                std::cout << i << " ";
            }
        }
        std::cout << std::endl;

    }
    return 0;
}
