#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void initializePriorities(float* priorities,hiprandState * d_state) {
    hiprand_init(1234, threadIdx.x, 0, &d_state[threadIdx.x]);
    priorities[threadIdx.x] = hiprand_uniform(d_state + threadIdx.x);
}

__global__ void checkMax(int* removed, int** graph,float* priorities,int* inMIS, int n){
    if (removed[threadIdx.x] == 1) return;
    bool isMaxPriority = true;
        for (int j = 0; j < n; ++j) {
            if (graph[threadIdx.x][j] == 1 && removed[j] == 0 && priorities[j] >= priorities[threadIdx.x]) {
                isMaxPriority = false;
                break;
            }
        }

        if (isMaxPriority) {
                inMIS[threadIdx.x] = 1;
        }
}

__global__ void removeVertices(int* removed, int** graph,int* inMIS,bool& changes, int n){
    if (inMIS[threadIdx.x] == 1) {
        removed[threadIdx.x] = 1;
        for (int j = 0; j < n; ++j) {
            if (graph[threadIdx.x][j] == 1) {
                removed[j] = 1;
            }
        }
        changes = true; // If any vertex is added, flag a change
    }
}


// Luby's Algorithm with Kokkos
int* lubysAlgorithm(int* removed, int** graph,float* priorities,int* inMIS, int n) {
    int* independentSet = new int[n];
    bool changes;
    bool d_changes;
    hipMalloc(&d_changes, sizeof(bool));
    hiprandState *d_state;
    hipMalloc(&d_state, sizeof(hiprandState));
    do {
        // Step 1: Assign random priorities to remaining vertices
        initializePriorities<<<1,n>>>(priorities,d_state);
        checkMax<<<1,n>>>(removed,graph,priorities,inMIS,n);
        // Step 3: Add selected vertices to MIS and remove them and their neighbors
        changes = false;
        hipMemcpy(d_changes,changes,sizeof(bool),hipMemcpyHostToDevice);
        removeVertices<<<1,n>>>(removed,graph,inMIS,changes,n);
        hipMemcpy(changes,d_changes,sizeof(bool),hipMemcpyDeviceToHost);
    } while (changes);
    hipMemcpy(independentSet,inMIS, n*sizeof(int), hipMemcpyDeviceToHost);
    return independentSet;
}

int main(int argc, char* argv[]) {
    {
        //Initialize graph
        int n = 6;
        int** adj = new int*[n];
        for(int i = 0;i < n; ++i){
            adj[i] = new int[n];
            for(int j = 0;j < n; ++j){
                adj[i][j] = 0;
            }
        }
        adj[0][1] = 1;
        adj[0][2] = 1;
        adj[1][3] = 1;
        adj[2][3] = 1;
        adj[3][4] = 1;
        adj[3][5] = 1;
        // Run Luby's algorithm with Kokkos
        int** d_adj;
        int* inMIS = new int[n];
        int* removed = new int[n];
        float* priorities = new float[n];
        int* independentSet = new int[n];
        hipMalloc(&inMIS,n*sizeof(int));
        hipMalloc(&removed,n*sizeof(int));
        hipMalloc(&priorities,n*sizeof(float));
        hipMalloc(&d_adj,n*n*sizeof(int));
        hipMemcpy(d_adj,adj,n*n*sizeof(int),hipMemcpyHostToDevice);
        independentSet = lubysAlgorithm(removed,adj,priorities,inMIS,n);
        hipFree(inMIS);
        hipFree(removed);
        hipFree(priorities);
        hipFree(d_adj);
        // Print the result
        std::cout << "Maximum Independent Set (MIS) nodes: " << std::endl;
        for(int i = 0; i < n; ++i){
            if (independentSet[i] == 1) {
                std::cout << i << " ";
            }
        }
        std::cout << std::endl;

    }
    return 0;
}
