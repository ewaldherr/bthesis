#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

__global__ void initializePriorities(float* priorities,hiprandState * d_state) {
    hiprand_init(1234, threadIdx.x, 0, &d_state[threadIdx.x]);
    priorities[threadIdx.x] = hiprand_uniform(d_state + threadIdx.x);
}

__global__ void checkMax(int** graph,float* priorities,int* state, int n){
    if (state[threadIdx.x] != 0) return;
    bool isMaxPriority = true;
        for (int j = 0; j < n; ++j) {
            if (graph[threadIdx.x][j] == 1 && state[j] == 0 && priorities[j] >= priorities[threadIdx.x]) {
                isMaxPriority = false;
                break;
            }
        }

        if (isMaxPriority) {
                state[threadIdx.x] = 1;
        }
}

__global__ void removeVertices( int** graph,int* state,bool* changes, int n){
    if (state[threadIdx.x] == 1) {
        state[threadIdx.x] = 2;
        for (int j = 0; j < n; ++j) {
            if (graph[threadIdx.x][j] == 1) {
                state[j] = -1;
            }
        }
        changes[0] = true; // If any vertex is added, flag a change
    }
}


// Luby's Algorithm with Kokkos
int* lubysAlgorithm(int** graph,float* priorities,int* state, int n) {
    int** host_adj;
    hipMemcpy(host_adj,graph,n*n*sizeof(int),hipMemcpyDeviceToHost);
    for (int i=0;i<n;++i){
        std::cout << std::endl;
        for(intj==;j<n;++j){
            std::cout << host_adj [i][j] << " ";
        }
    }
    std::cout << std::endl;
    float* host_prios = new float[n];
    int* host_state = new int[n];
    int* independentSet = new int[n];
    bool* changes = new bool[1];
    bool* d_changes = new bool[1];
    hipMalloc(&d_changes, sizeof(bool));
    hiprandState *d_state;
    hipMalloc(&d_state, sizeof(hiprandState));
    int iters = 0;
    do {
        // Step 1: Assign random priorities to remaining vertices
        initializePriorities<<<1,n>>>(priorities,d_state);
        hipMemcpy(host_prios,priorities,n*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(host_state,state,n*sizeof(int),hipMemcpyDeviceToHost);
        for (int i = 0; i< n; ++i){
            	std::cout << host_prios[i] << " " << host_state[i] << "  ";
        }
        std::cout << std::endl;
        checkMax<<<1,n>>>(graph,priorities,state,n);
        hipMemcpy(host_state,state,n*sizeof(int),hipMemcpyDeviceToHost);
        for (int i = 0; i< n; ++i){
            	std::cout << host_prios[i] << " " << host_state[i] << "  ";
        }
        std::cout << std::endl;
        // Step 3: Add selected vertices to MIS and remove them and their neighbors
        changes[0] = false;
        hipMemcpy(d_changes,changes,sizeof(bool),hipMemcpyHostToDevice);
        removeVertices<<<1,n>>>(graph,state,changes,n);
        hipMemcpy(host_state,state,n*sizeof(int),hipMemcpyDeviceToHost);
        for (int i = 0; i< n; ++i){
            	std::cout << host_prios[i] << " " << host_state[i] << "  ";
        }
        std::cout << std::endl;
        hipMemcpy(changes,d_changes,sizeof(bool),hipMemcpyDeviceToHost);
        ++ iters;
    } while (changes[0]);
    std::cout << iters << std::endl;
    hipMemcpy(independentSet,state, n*sizeof(int), hipMemcpyDeviceToHost);
    return independentSet;
}

int main(int argc, char* argv[]) {
    {
        //Initialize graph
        int n = 6;
        int** adj = new int*[n];
        for(int i = 0;i < n; ++i){
            adj[i] = new int[n];
            for(int j = 0;j < n; ++j){
                adj[i][j] = 0;
            }
        }
        adj[0][1] = 1;
        adj[0][2] = 1;
        adj[1][3] = 1;
        adj[2][3] = 1;
        adj[3][4] = 1;
        adj[3][5] = 1;
        //backward edges
        adj[1][0] = 1;
        adj[2][0] = 1;
        adj[3][1] = 1;
        adj[3][2] = 1;
        adj[4][3] = 1;
        adj[5][3] = 1;
        // Run Luby's algorithm with Kokkos
        int** d_adj;
        int* host_state = new int[n];
        for(int i = 0; i < n; ++i){
            host_state[i] = 0;
        }
        int* state = new int[n];
        float* priorities = new float[n];
        int* independentSet = new int[n];
        hipMalloc(&state,n*sizeof(int));
        hipMalloc(&state,n*sizeof(int));
        hipMalloc(&priorities,n*sizeof(float));
        hipMalloc(&d_adj,n*n*sizeof(int));
        hipMemcpy(state,host_state,n*sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(d_adj,adj,n*n*sizeof(int),hipMemcpyHostToDevice);
        independentSet = lubysAlgorithm(adj,priorities,state,n);
        hipFree(state);
        hipFree(priorities);
        hipFree(d_adj);
        // Print the result
        std::cout << "Maximum Independent Set (MIS) nodes: " << std::endl;
        for(int i = 0; i < n; ++i){
            if (independentSet[i] == 2) {
                std::cout << i << " ";
            }
        }
        std::cout << std::endl;

    }
    return 0;
}
